#include "hip/hip_runtime.h"
#include <algorithm>
#include <stdio.h>
#include <stdlib.h>

#include "stage3.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

DECLARE_string(target);
DECLARE_bool(runtests);
DEFINE_string(input_shard, "target.out.0",
              "The filename of the stage1 shard to run on.");
DECLARE_string(output);
DECLARE_int32(srand_seed);
DEFINE_int32(stop_after, -1,
             "If set to a positive value, the program "
             "will stop after processing <stop_after> stage1 candidates.");
DEFINE_int32(cuda_device, -1,
             "Which CUDA device to use, -1 to use them all.");
DECLARE_int32(test_archive);
DEFINE_int32(test_archive, 0,
             "Which test archive to use when running tests.");

using namespace mitm;
using namespace mitm_stage1;
using namespace mitm_stage2;
using namespace stage3;
using namespace std;
using namespace breakzip;
using namespace google;

const char *usage_message = R"usage(
    Usage: mitm_stage2 <FILE> <OUT>
    Runs the stage2 attack using the stage1 data in FILE, the shard specified
    by -shard, and writes output to the filename specified by -outfile with the
    shard number appended.

    If you pass the -runtests argument, then the tests will fail unless the
    correct guess is contained within the shard you have provided via -target.
    Stage1 prints the name of the shard containing the correct guess.
    )usage";

// Print device properties
void print_device_properties(hipDeviceProp_t devProp) {
    printf("Major revision number:         %d\n",  devProp.major);
    printf("Minor revision number:         %d\n",  devProp.minor);
    printf("Name:                          %s\n",  devProp.name);
    printf("Total global memory:           %lu\n",  devProp.totalGlobalMem);
    printf("Total shared memory per block: %lu\n",  devProp.sharedMemPerBlock);
    printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
    printf("Warp size:                     %d\n",  devProp.warpSize);
    printf("Maximum memory pitch:          %lu\n",  devProp.memPitch);
    printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
    printf("Maximum threads per multiproc: %d\n", devProp.maxThreadsPerMultiProcessor);

    for (int i = 0; i < 3; ++i) {
        printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
    }

    for (int i = 0; i < 3; ++i) {
        printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
    }

    printf("Clock rate:                    %d\n",  devProp.clockRate);
    printf("Total constant memory:         %lu\n",  devProp.totalConstMem);
    printf("Texture alignment:             %lu\n",  devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
    printf("\n\n");
    return;
}

__global__ void gpu_stage3_kernel(const gpu_stage2_candidate *candidates,
                                  keys *results,
                                  const archive_info* archive,
                                  const uint32_t stage2_candidate_count,
                                  const mitm::correct_guess& c) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < stage2_candidate_count) {
        keys result = {0, 0, 0};
        stage3::gpu_stage3_internal(*archive, candidates[i], &result, &c);
        results[i].crck00 = result.crck00;
        results[i].k10 = result.k10;
        results[i].k20 = result.k20;
    }
}

int main(int argc, char *argv[]) {
    int my_argc = argc;

    SetVersionString(version_string());
    SetUsageMessage(usage_message);
    auto non_flag = ParseCommandLineFlags(&my_argc, &argv, false);

    // We build the preimages once for all candidates.
    vector<vector<uint16_t>> preimages(0x100);
    build_preimages(preimages);

    gpu_stage2_candidate *stage2_candidates = nullptr;
    uint32_t stage2_candidate_count = 0;
    size_t candidate_array_size = 0;
    read_stage2_candidates_for_gpu(&stage2_candidates, &stage2_candidate_count, candidate_array_size);
    size_t results_array_size = sizeof(keys) * stage2_candidate_count;

    if (0 == stage2_candidate_count) {
        fprintf(stderr, "FATAL: Read no candidates from input file.\n");
        exit(-1);
    }

    if (nullptr == stage2_candidates) {
        fprintf(stderr, "FATAL: Stage2 candidate array was null.\n");
        exit(-1);
    }

    fprintf(stdout, "Read %d candidates from stage2.\n",
            stage2_candidate_count);

    archive_info archive;
    correct_guess guess[2] = {correct(mitm::test[FLAGS_test_archive])};
    correct_guess *c = nullptr;

    // Generate the x array from the seed.
    srand(FLAGS_srand_seed);
    for (int j = 0; j < 2; ++j) {
        for (int i = 0; i < 10; ++i) {
            archive.file[j].x[i] = rand() >> 7;
        }
    }

    // Acquire the h array from the file.
    auto zfile = new ZipFile(FLAGS_target);
    if (0 != zfile->init()) {
        perror("Couldn't initialize target ZIP file");
        exit(-1);
    }

    auto lfhs = zfile->local_file_headers();
    // NB(leaf): This is a bug if the target file has more than two files
    // because the MITM types don't support more than two.
    for (int i = 0; i < lfhs.size(); ++i) {
        auto crypt_header = lfhs[i]->crypt_header();
        for (int j = 0; j < 10; ++j) {
            archive.file[i].h[j] = crypt_header[j];
        }
    }

    if (FLAGS_runtests) {
        c = &(guess[0]);
        archive = mitm::test[FLAGS_test_archive];
    }

    if ((archive.file[0].x[0] != archive.file[0].h[0]) ||
        (archive.file[1].x[0] != archive.file[1].h[0])) {
        fprintf(stderr, "Given seed does not generate the initial bytes!");
        exit(-1);
    }

    int cuda_device_count = 0;
    hipGetDeviceCount(&cuda_device_count);

    if (0 == cuda_device_count) {
        fprintf(stderr, "Host has no CUDA capable devices. Use cpu_stage3, instead?\n");
        exit(-1);
    }

    vector<int> target_devices;
    int cuda_device = 0;
    for (cuda_device = 0; cuda_device < cuda_device_count; ++cuda_device) {
        struct hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, cuda_device);
        fprintf(stderr, "CUDA Device %d: %s\n", cuda_device, prop.name);
        print_device_properties(prop);
        
        // TODO(leaf): Check capabilities for what we need?
        if (-1 == FLAGS_cuda_device) {
            target_devices.push_back(cuda_device);
            fprintf(stderr, "Targeting CUDA device %d\n", cuda_device);
        } else if (FLAGS_cuda_device == cuda_device) {
            target_devices.push_back(cuda_device);
            fprintf(stderr, "stage3 will target CUDA device %d\n", cuda_device);
        } else {
            fprintf(stderr, "Ignoring CUDA device %d\n", cuda_device);
        }

        fprintf(stderr, "\n");
    }

    fprintf(stderr, "CUDA stage3 targeting these devices: ");
    for_each(target_devices.begin(), target_devices.end(),
             [](const auto &e) { fprintf(stderr, "%d ", e); });
    fprintf(stderr, "\n");

    for (auto device: target_devices) {
        fprintf(stderr, "Initialization device %d...\n", device);
        auto err = hipSetDevice(device);
        if (hipSuccess != err) {
            fprintf(stderr, "Failed to set active CUDA device %d: %s\n",
                    device, hipGetErrorString(err));
            exit(-1);
        }

        // Allocate host memory for results
        keys *host_results = (keys *)::calloc(stage2_candidate_count, sizeof(keys));
        if (nullptr == host_results) {
            fprintf(stderr, "Failed to allocate host memory for result set.\n");
            exit(-1);
        }

        archive_info *dev_archive = nullptr;
        err = hipMalloc(&dev_archive, sizeof(archive_info));
        if (hipSuccess != err) {
            fprintf(stderr, "Failed to allocate memory for archive on device %d\n", device);
            exit(-1);
        }
        err = hipMemcpy(dev_archive, &archive, sizeof(archive_info), hipMemcpyHostToDevice);
        if (hipSuccess != err) {
            fprintf(stderr, "Failed to memcpy archive to CUDA device %d\n", device);
            exit(-1);
        }

        // Allocate device memory
        gpu_stage2_candidate *dev_cands = nullptr;
        keys *dev_results = nullptr;
        fprintf(stderr, "Allocating candidate array of size %ld on device %d\n",
                candidate_array_size, device);
        err = hipMalloc(&dev_cands, candidate_array_size);
        if (hipSuccess != err) {
            fprintf(stderr, "Failed to allocate memory on CUDA device %d: %s\n",
                    device, hipGetErrorString(err));
            exit(-1);
        }

        if (nullptr == dev_cands) {
            fprintf(stderr, "Device allocation failed, array is null!\n");
            exit(-1);
        }

        fprintf(stderr, "Allocating results array of size %ld on device %d\n", results_array_size, device);
        err = hipMalloc(&dev_results, results_array_size);
        if (hipSuccess != err) {
            fprintf(stderr, "Failed to allocate memory on CUDA device %d: %s\n",
                    device, hipGetErrorString(err));
            hipFree(dev_cands);
            exit(-1);
        }

        // Copy candidates to device
        fprintf(stderr, "Copying candidate data to device %d\n", device);
        err = hipMemcpy(dev_cands, stage2_candidates, candidate_array_size, hipMemcpyHostToDevice);
        if (hipSuccess != err) {
            fprintf(stderr, "Failed to memcpy data to CUDA device %d: %s\n",
                    device, hipGetErrorString(err));
            exit(-1);
        }

        // Call kernel
        fprintf(stderr, "Calling kernel:\n  Host results @ %p\n  Dev  results @ %p\n  Results size: %ld\n",
                host_results, dev_results, results_array_size);
        fprintf(stderr, "  Dec candidates @ %p\n  Stage2 candidates: %u\n", dev_cands, stage2_candidate_count);

        int block_size = 0;
        int min_grid_size = 0;
        int grid_size = 0;

        hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, (void*)gpu_stage3_kernel, 0, stage2_candidate_count);

        fprintf(stderr, "Est.   BlockSize: block_sz=%d min_grid_sz=%d grid_size=%d\n", block_size, min_grid_size, grid_size);
        //round up
        grid_size = (stage2_candidate_count + block_size - 1) / block_size;
        fprintf(stderr, "Actual BlockSize: block_sz=%d min_grid_sz=%d grid_size=%d\n", block_size, min_grid_size, grid_size);


        gpu_stage3_kernel<<<grid_size, block_size>>>(dev_cands, dev_results, dev_archive, stage2_candidate_count, *c);
        err = hipGetLastError();
        if (hipSuccess != err) {
            fprintf(stderr, "CUDA Kernel failed: %s\n", hipGetErrorString(err));
            hipFree(dev_cands);
            hipFree(dev_results);
            free(host_results);
            exit(-1);
        }

        // Copy results to host.
        err = hipMemcpy(host_results, dev_results, results_array_size, hipMemcpyDeviceToHost);
        if (hipSuccess != err) {
            fprintf(stderr, "Failed to copy results to host: %s\n",
                    hipGetErrorString(err));
            fprintf(stderr, "  Host results @ %p\n", host_results);
            fprintf(stderr, "  Dev  results @ %p\n", dev_results);
            fprintf(stderr, "  Results size: %ld\n", results_array_size);
            hipFree(dev_cands);
            hipFree(dev_results);
            free(host_results);
            exit(-1);
        }

        bool success = false;
        for (int i = 0; i < stage2_candidate_count; ++i) {
            if (host_results[i].crck00 != 0 || host_results[i].k10 != 0 || host_results[i].k20 != 0) {
                fprintf(stdout, "FINAL: Success! Keys: crck00=%08x k10=%08x k20=%08x\n",
                        host_results[i].crck00,
                        host_results[i].k10,
                        host_results[i].k20);
                auto keyfile = fopen(FLAGS_output.c_str(), "a+");
                if (nullptr == keyfile) {
                    fprintf(stderr, "Can't open output key file %s: %s",
                            FLAGS_output.c_str(), strerror(errno));
                } else {
                    fprintf(keyfile, "Valid Keys Found: crck00=%08x k10=%08x k20=%08x\n",
                            host_results[i].crck00,
                            host_results[i].k10,
                            host_results[i].k20);
                    fclose(keyfile);
                }

                success = true;
                break;
            }
        }

        if (!success) {
            fprintf(stderr, "FINAL: Results check complete, no keys found.\n");
        }

        // Free memory on device.
        err = hipFree(dev_cands);
        if (hipSuccess != err) {
            fprintf(stderr, "Failed to free device memory on %d: %s\n",
                    device, hipGetErrorString(err));
            exit(-1);
        }

        err = hipFree(dev_results);
        if (hipSuccess != err) {
            fprintf(stderr, "Failed to free device memory on %d: %s\n",
                    device, hipGetErrorString(err));
            exit(-1);
        }

        err = hipFree(dev_archive);
        if (hipSuccess != err) {
            fprintf(stderr, "Failed to free device memory on %d: %s\n",
                    device, hipGetErrorString(err));
            exit(-1);
        }
    }

    exit(0);
}
