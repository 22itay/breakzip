#include "hip/hip_runtime.h"
/* Copyright (c) 2016, Pyrofex Corporation.
 * All right reserved.
 * Author: Nash E. Foster <leaf@pyrofex.com>
 */
#include <iostream>
#include <stdlib.h>
#include <check.h>
#include <string>

#include <gflags/gflags.h>

#include "../breakzip.h"

using namespace breakzip;
using namespace std;

#include <stdio.h>
#include <cassert>
#include <hip/hip_runtime.h>
#include <sys/utsname.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

const char *sampleName = "simpleAssert";

__device__ __managed__ int ret[1000];
__global__ void testKernel(int a, int b) {
    ret[threadIdx.x] = a + b + threadIdx.x;
}

START_TEST(test_always_pass) {
    findCudaDevice(0, nullptr);
    int A = 10, B = 100;
    testKernel<<<1, 1000>>>(A, B);
    hipDeviceSynchronize();

    for (int i = 0; i < 1000; i++) {
        printf("%d: A+B = %d\n", i, ret[i]);
        ck_assert(ret[i] == (A + B + i));
    }
}
END_TEST


Suite* make_suite(const std::string name) {
    Suite* s;
    TCase* tc_core;
    s = suite_create(name.c_str());
    tc_core = tcase_create("core");

    /* Add every test case that you write below here. */
    tcase_add_test(tc_core, test_always_pass);

    suite_add_tcase(s, tc_core);
    return s;
}

int main(int argc, char* argv[]) {
    vector<string> inputs;
    
    int failed = 0;
    Suite* s = nullptr;
    SRunner* sr = nullptr;
    s = make_suite("pyr8::TEMPLATE");
    sr = srunner_create(s);
    srunner_run_all(sr, CK_VERBOSE);
    failed = srunner_ntests_failed(sr);
    srunner_free(sr);

    auto return_val = (0 == failed ? EXIT_SUCCESS : EXIT_FAILURE);
    return return_val;
}
